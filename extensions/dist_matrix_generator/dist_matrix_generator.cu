#include "hip/hip_runtime.h"
/*
 * @Author: Haozhe Xie
 * @Date:   2020-08-14 17:22:12
 * @Last Modified by:   Haozhe Xie
 * @Last Modified time: 2020-09-18 15:02:44
 * @Email:  cshzxie@gmail.com
 */

#include <iostream>
#include <torch/types.h>

#define CUDA_NUM_THREADS 512

__global__ void dist_matrix_generator_cuda_kernel(
    int n_objects, int height, int width, const float prob_threshold,
    const float occ_dist_factor, const float *__restrict__ mask,
    const float *__restrict__ occ_mask, int *__restrict__ bboxes,
    int *__restrict__ n_points, float *__restrict__ dist_matrix) {
  int batch_index = blockIdx.x;
  int thread_index = threadIdx.x;
  int stride = blockDim.x;
  int n_pixels = height * width;

  bboxes += batch_index * n_objects * 4;
  n_points += batch_index * n_objects;
  mask += batch_index * n_objects * n_pixels;
  dist_matrix += batch_index * n_objects * n_pixels;

  // Initialize the values for bboxes
  for (int i = 1; i < n_objects; ++i) {
    bboxes[i * 4] = 32767;
    bboxes[i * 4 + 2] = 32767;
  }

  // Get the bounding boxes of objects
  for (int i = 1; i < n_objects; ++i) {
    for (int j = thread_index; j < n_pixels; j += stride) {
      int x = j % width;
      int y = j / width;

      if (mask[i * n_pixels + j] >= prob_threshold) {
        atomicAdd(&n_points[i], 1);
        atomicMin(&bboxes[i * 4], x);     // bbox: x_min
        atomicMax(&bboxes[i * 4 + 1], x); // bbox: x_max
        atomicMin(&bboxes[i * 4 + 2], y); // bbox: y_min
        atomicMax(&bboxes[i * 4 + 3], y); // bbox: y_max
      }
    }
  }
  __syncthreads();

  // Calculate the distance matrix according to the bounding boxes
  for (int i = 1; i < n_objects; ++i) {
    if (n_points[i] == 0) {
      continue;
    }

    for (int j = thread_index; j < n_pixels; j += stride) {
      int x = j % width;
      int y = j / width;
      float diff_x = 0;
      float diff_y = 0;

      if (x < bboxes[i * 4]) {
        diff_x = bboxes[i * 4] - x;
      } else if (x > bboxes[i * 4 + 1]) {
        diff_x = bboxes[i * 4 + 1] - x;
      }
      if (y < bboxes[i * 4 + 2]) {
        diff_y = bboxes[i * 4 + 2] - y;
      } else if (y > bboxes[i * 4 + 3]) {
        diff_y = bboxes[i * 4 + 3] - y;
      }

      // Determine the values of the distance matrix
      float dist = diff_x * diff_x + diff_y * diff_y;
      dist_matrix[i * n_pixels + j] = dist;
      // Determine the values of occluded regions according to the bounding
      // boxes
      if (occ_mask[i * n_pixels + j] < 0.5) {
        dist_matrix[i * n_pixels + j] = dist * occ_dist_factor;
      }
    }
  }
}

torch::Tensor dist_matrix_generator_cuda_forward(torch::Tensor mask,
                                                 torch::Tensor occ_mask,
                                                 float prob_threshold,
                                                 float occ_dist_factor,
                                                 hipStream_t stream) {
  int batch_size = mask.size(0);
  int n_objects = mask.size(1);
  int height = mask.size(2);
  int width = mask.size(3);

  torch::Tensor bboxes =
      torch::zeros({batch_size, n_objects, 4}, torch::CUDA(torch::kInt));
  torch::Tensor n_points =
      torch::zeros({batch_size, n_objects}, torch::CUDA(torch::kInt));
  torch::Tensor dist_matrix = torch::zeros(
      {batch_size, n_objects, height, width}, torch::CUDA(torch::kFloat));

  dist_matrix_generator_cuda_kernel<<<batch_size, CUDA_NUM_THREADS, 0,
                                      stream>>>(
      n_objects, height, width, prob_threshold, occ_dist_factor,
      mask.data_ptr<float>(), occ_mask.data_ptr<float>(),
      bboxes.data_ptr<int>(), n_points.data_ptr<int>(),
      dist_matrix.data_ptr<float>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Error in dist_matrix_generator_cuda_forward: "
              << hipGetErrorString(err);
  }
  return dist_matrix;
}
