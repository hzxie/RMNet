#include "hip/hip_runtime.h"
/*
 * @Author: Haozhe Xie
 * @Date:   2020-08-14 17:22:12
 * @Last Modified by:   Haozhe Xie
 * @Last Modified time: 2020-11-01 14:35:20
 * @Email:  cshzxie@gmail.com
 */

#include <iostream>
#include <cmath>
#include <torch/types.h>

#define CUDA_NUM_THREADS 512

__global__ void reg_att_map_generator_cuda_kernel(
    int n_objects, int height, int width, const float prob_threshold,
    int n_pts_threshold, int dist_threshold, const float *__restrict__ mask,
    int *__restrict__ bboxes, int *__restrict__ n_points,
    float *__restrict__ reg_att_map) {
  int batch_index = blockIdx.x;
  int thread_index = threadIdx.x;
  int stride = blockDim.x;
  int n_pixels = height * width;

  bboxes += batch_index * n_objects * 4;
  n_points += batch_index * n_objects;
  mask += batch_index * n_objects * n_pixels;
  reg_att_map += batch_index * n_objects * n_pixels;

  // Initialize the values for bboxes
  for (int i = 1; i < n_objects; ++i) {
    bboxes[i * 4] = 32767;
    bboxes[i * 4 + 2] = 32767;
  }

  // Get the bounding boxes of objects
  for (int i = 1; i < n_objects; ++i) {
    for (int j = thread_index; j < n_pixels; j += stride) {
      int x = j % width;
      int y = j / width;

      if (mask[i * n_pixels + j] >= prob_threshold) {
        atomicAdd(&n_points[i], 1);
        atomicMin(&bboxes[i * 4], x);     // bbox: x_min
        atomicMax(&bboxes[i * 4 + 1], x); // bbox: x_max
        atomicMin(&bboxes[i * 4 + 2], y); // bbox: y_min
        atomicMax(&bboxes[i * 4 + 3], y); // bbox: y_max
      }
    }
  }
  __syncthreads();

  // Calculate the distance matrix according to the bounding boxes
  for (int i = 1; i < n_objects; ++i) {
    if (n_points[i] < n_pts_threshold) {
      continue;
    }

    for (int j = thread_index; j < n_pixels; j += stride) {
      int x = j % width;
      int y = j / width;
      float diff_x = 0;
      float diff_y = 0;

      if (x < bboxes[i * 4]) {
        diff_x = bboxes[i * 4] - x;
      } else if (x > bboxes[i * 4 + 1]) {
        diff_x = bboxes[i * 4 + 1] - x;
      }
      if (y < bboxes[i * 4 + 2]) {
        diff_y = bboxes[i * 4 + 2] - y;
      } else if (y > bboxes[i * 4 + 3]) {
        diff_y = bboxes[i * 4 + 3] - y;
      }

      // Determine the values of the attention map
      float dist = std::sqrt(diff_x * diff_x + diff_y * diff_y);
      if (dist > dist_threshold) {
        reg_att_map[i * n_pixels + j] = 0;
      }
    }
  }
}

torch::Tensor reg_att_map_generator_cuda_forward(torch::Tensor mask,
                                                 float prob_threshold,
                                                 int n_pts_threshold,
                                                 int dist_threshold,
                                                 hipStream_t stream) {
  int batch_size = mask.size(0);
  int n_objects = mask.size(1);
  int height = mask.size(2);
  int width = mask.size(3);

  torch::Tensor bboxes =
      torch::zeros({batch_size, n_objects, 4}, torch::CUDA(torch::kInt));
  torch::Tensor n_points =
      torch::zeros({batch_size, n_objects}, torch::CUDA(torch::kInt));
  torch::Tensor reg_att_map = torch::ones(
      {batch_size, n_objects, height, width}, torch::CUDA(torch::kFloat));

  reg_att_map_generator_cuda_kernel<<<batch_size, CUDA_NUM_THREADS, 0,
                                      stream>>>(
      n_objects, height, width, prob_threshold, n_pts_threshold, dist_threshold,
      mask.data_ptr<float>(), bboxes.data_ptr<int>(), n_points.data_ptr<int>(),
      reg_att_map.data_ptr<float>());

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cout << "Error in reg_att_map_generator_cuda_forward: "
              << hipGetErrorString(err);
  }
  return reg_att_map;
}
